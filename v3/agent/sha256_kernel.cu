#include "hip/hip_runtime.h"
/******************************************************************************
*                                                                             *
* Distributed Hash Cracker v3.0                                               *
*                                                                             *
* Copyright (c) 2009 RPISEC.                                                  *
* All rights reserved.                                                        *
*                                                                             *
* SHA256 Implementation Copyright (C) 2010 Samuel Rodriguez-Sevilla           *
*                                                                             *
* Redistribution and use in source and binary forms, with or without modifi-  *
* cation, are permitted provided that the following conditions are met:       *
*                                                                             *
*    * Redistributions of source code must retain the above copyright notice  *
*      this list of conditions and the following disclaimer.                  *
*                                                                             *
*    * Redistributions in binary form must reproduce the above copyright      *
*      notice, this list of conditions and the following disclaimer in the    *
*      documentation and/or other materials provided with the distribution.   *
*                                                                             *
*    * Neither the name of RPISEC nor the names of its contributors may be    *
*      used to endorse or promote products derived from this software without *
*      specific prior written permission.                                     *
*                                                                             *
* THIS SOFTWARE IS PROVIDED BY RPISEC "AS IS" AND ANY EXPRESS OR IMPLIED      *
* WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF        *
* MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN     *
* NO EVENT SHALL RPISEC BE HELD LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,  *
* SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED    *
* TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR      *
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF      *
* LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING        *
* NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS          *
* SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.                *
*                                                                             *
******************************************************************************/


/*!
	@file sha256_kernel.cu
	
	@brief CUDA implementation of SHA256
 */

//Textures
texture<int, 1, hipReadModeElementType> texCharset;

#define InitGuess(num, a,b,c,d)				\
		{										\
			/* Calculate the four indices */	\
			LstartInit(lstart3, a);				\
			LstartInit(lstart2, b);				\
			LstartInit(lstart1, c);				\
			LstartInit(lstart0, d);				\
			/* Pack four elements into the int (if we exceed length, padding will overwrite the garbage) */	\
			w##num = 							\
			(charset[lstart3] << 24) | 			\
			(charset[lstart2] << 16) | 			\
			(charset[lstart1] << 8) |  			\
			 charset[lstart0];					\
		}
		 
#define LstartInit(ls, num)								\
		{												\
			/* Get initial value and apply carry-in */	\
			ls = carry + start[num]; 					\
			/* Rightmost value? Bump by index */		\
			if(num == lm1) 								\
				ls += index;							\
			/* Carry out */								\
			if(ls >= base && num<len)					\
			{											\
				/* Calculate carry */					\
				carry = ls / base;						\
				/* Update this digit */					\
				ls %= base;								\
			}											\
			else										\
				carry = 0;								\
		}


/*!
	@brief CUDA implementation of SHA256	
	Thread-per-block requirement: minimum 64
	
	@param gtarget Target value (five ints, little endian)
	@param gstart Start index in charset (array of 32 ints, data is left aligned, unused values are at right)
	@param gsalt Salt (not used)
	@param status Set to true by a thread which succeeds in cracking the hash
	@param output Set to the collision by a thread which succeeds in cracking the hash
	@param base Length of the character set (passed in texCharset texture)
	@param len Length of valid data in gstart
	@param saltlen Length of salt (not used)
	@param hashcount Number of hashes being tested (not used)
 */
extern "C" __global__ void sha256Kernel(int* gtarget, int* gstart, char* gsalt, char* status, char* output, int base, int len, int saltlen, int hashcount)
{
	//Get our position in the grid
	int index = (blockDim.x * blockIdx.x) + threadIdx.x;
	
	//Cache charset in shmem
	__shared__ char charset[256];
	if(threadIdx.x < ceil((float)base / 4))
	{
		int* ccs = (int*)&charset[0];
		ccs[threadIdx.x] = tex1Dfetch(texCharset, threadIdx.x);
	}
	
	//Cache start value
	__shared__ int start[32];
	if(threadIdx.x < len)
		start[threadIdx.x] = gstart[threadIdx.x];
	
	//Cache target value (do byte-swap here)
	__shared__ int target[8];
	if(threadIdx.x < 8)
		target[threadIdx.x] = bswap(gtarget[threadIdx.x]);
	
	//Wait for all cache filling to finish
	__syncthreads();
	
	//Core round functions
	#include "sha256_kernel_core.h"
		
	//Test the output
	int* pt = (int*)target;
	if( (pt[0] == a) && (pt[1] == b) && (pt[2] == c) && (pt[3] == d) && (pt[4] == e) && (pt[5] == f) && (pt[6] == g) && (pt[7] == h))
	{
		//If we get here, we must be a match! Save the result and quit
		*status = 1;
		unsigned int lo4 = len/4;
		
		unsigned int lstart0=0, lstart1=0, lstart2=0, lstart3=0;
		unsigned int lm1 = len-1;
		unsigned int carry = 0;		//no initial carry-in
		switch(lo4)
		{
		case 7:
			InitGuess(7, 31,30,29,28);
		case 6:
			InitGuess(6, 27,26,25,24);
		case 5:
			InitGuess(5, 23,22,21,20);
		case 4:
			InitGuess(4, 19,18,17,16);
		case 3:
			InitGuess(3, 15,14,13,12);
		case 2:
			InitGuess(2, 11,10,9,8);
		case 1:
			InitGuess(1, 7,6,5,4);
		case 0:
		default:
			InitGuess(0, 3,2,1,0);
		}
		
		switch(lo4)
		{
		SaveOutput(7);
		SaveOutput(6);
		SaveOutput(5);
		SaveOutput(4);
		SaveOutput(3);	
		SaveOutput(2);
		SaveOutput(1);
		SaveOutput(0);
		}
	}
}
