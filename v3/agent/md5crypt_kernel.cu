#include "hip/hip_runtime.h"
/******************************************************************************
*                                                                             *
* Distributed Hash Cracker v3.0                                               *
*                                                                             *
* Copyright (c) 2009 RPISEC.                                                  *
* All rights reserved.                                                        *
*                                                                             *
* Redistribution and use in source and binary forms, with or without modifi-  *
* cation, are permitted provided that the following conditions are met:       *
*                                                                             *
*    * Redistributions of source code must retain the above copyright notice  *
*      this list of conditions and the following disclaimer.                  *
*                                                                             *
*    * Redistributions in binary form must reproduce the above copyright      *
*      notice, this list of conditions and the following disclaimer in the    *
*      documentation and/or other materials provided with the distribution.   *
*                                                                             *
*    * Neither the name of RPISEC nor the names of its contributors may be    *
*      used to endorse or promote products derived from this software without *
*      specific prior written permission.                                     *
*                                                                             *
* THIS SOFTWARE IS PROVIDED BY RPISEC "AS IS" AND ANY EXPRESS OR IMPLIED      *
* WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF        *
* MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN     *
* NO EVENT SHALL RPISEC BE HELD LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,  *
* SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED    *
* TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR      *
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF      *
* LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING        *
* NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS          *
* SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.                *
*                                                                             *
******************************************************************************/

/*!
	@file md5crypt_kernel.cu
	
	@brief CUDA implementation of MD5crypt
 */

//Textures
texture<int, 1, hipReadModeElementType> texCharset;

//Left rotate
#define ROTL(x, n) (((x) << (n)) | ((x) >> (32 - (n))))

//Core function
#define md5round_core(a,b,c,d,k,s,t,f) a = ROTL(a + f(b,c,d) + buf[k] + t,s) + b;
#define fastmd5round_core(a,b,c,d,k,s,t,f) a = ROTL(a + f(b,c,d) + k + t,s) + b;

//Round ops
#define Ff(b,c,d) (((b) & (c)) | (~(b) & (d)))
#define Fg(b,c,d) (((b) & (d)) | (~(d) & (c)))
#define Fh(b,c,d) ((b) ^ (c) ^ (d))
#define Fi(b,c,d) ((c) ^ ((b) | ~(d)))

//Rounds
#define md5round_f(a,b,c,d,i,n,t) md5round_core(a,b,c,d,i,n,t,Ff)
#define md5round_g(a,b,c,d,i,n,t) md5round_core(a,b,c,d,i,n,t,Fg)
#define md5round_h(a,b,c,d,i,n,t) md5round_core(a,b,c,d,i,n,t,Fh)
#define md5round_i(a,b,c,d,i,n,t) md5round_core(a,b,c,d,i,n,t,Fi)

#define fastmd5round_f(a,b,c,d,i,n,t) fastmd5round_core(a,b,c,d,i,n,t,Ff)
#define fastmd5round_g(a,b,c,d,i,n,t) fastmd5round_core(a,b,c,d,i,n,t,Fg)
#define fastmd5round_h(a,b,c,d,i,n,t) fastmd5round_core(a,b,c,d,i,n,t,Fh)
#define fastmd5round_i(a,b,c,d,i,n,t) fastmd5round_core(a,b,c,d,i,n,t,Fi)

void __device__ DoMD5Hash(unsigned int* pin, unsigned int* pout, int length)
{
	unsigned char* out = reinterpret_cast<unsigned char*>(pout);
	
	//////////////////////////////////////////////////////////////////////////////
	//STEP 1: Copy buffer and append padding bits
	//Generate the string
	unsigned int buf[16];								//Our computation buffer
	for(int j=floor(length/4.0f); j<16; j++)			//Zero the end
		buf[j] = 0;
	for(int j=0; j<ceilf(length/4.0f); j++)				//Copy string
		buf[j] = pin[j];
	unsigned char* p=(unsigned char*)&buf[0];			//Byte ptr to buffer
	p[length]=0x80;										//Add padding
	for(int i=length+1; i%4 != 0; i++)
		p[i] = 0;
	
	//////////////////////////////////////////////////////////////////////////////
	//STEP 2: Append Length
	buf[14]=length * 8;

	//////////////////////////////////////////////////////////////////////////////
	//STEP 3: Initialize Constants
	//Variable declarations
	unsigned int a,b,c,d;								//Temp variables for MD5 computation
	a = 0x67452301;
	b = 0xefcdab89;
	c = 0x98badcfe;
	d = 0x10325476;

	//////////////////////////////////////////////////////////////////////////////
	//STEP 4: MD5 Rounds
	md5round_f(a,b,c,d,0,7,0xd76aa478);
		md5round_f(d,a,b,c,1,12,0xe8c7b756);
		md5round_f(c,d,a,b,2,17,0x242070db);
		md5round_f(b,c,d,a,3,22,0xc1bdceee);
	md5round_f(a,b,c,d,4,7,0xf57c0faf);
		md5round_f(d,a,b,c,5,12,0x4787c62a);
		md5round_f(c,d,a,b,6,17,0xa8304613);
		md5round_f(b,c,d,a,7,22,0xfd469501);
	md5round_f(a,b,c,d,8,7,0x698098d8);
		md5round_f(d,a,b,c,9,12,0x8b44f7af);
		md5round_f(c,d,a,b,10,17,0xffff5bb1);
		md5round_f(b,c,d,a,11,22,0x895cd7be);
	md5round_f(a,b,c,d,12,7,0x6b901122);
		md5round_f(d,a,b,c,13,12,0xfd987193);
		md5round_f(c,d,a,b,14,17,0xa679438e);
		md5round_f(b,c,d,a,15,22,0x49b40821);

	md5round_g(a,b,c,d,1,5,0xf61e2562);
		md5round_g(d,a,b,c,6,9,0xc040b340);
		md5round_g(c,d,a,b,11,14,0x265e5a51);
		md5round_g(b,c,d,a,0,20,0xe9b6c7aa);
	md5round_g(a,b,c,d,5,5,0xd62f105d);
		md5round_g(d,a,b,c,10,9,0x02441453);
		md5round_g(c,d,a,b,15,14,0xd8a1e681);
		md5round_g(b,c,d,a,4,20,0xe7d3fbc8);
	md5round_g(a,b,c,d,9,5,0x21e1cde6);
		md5round_g(d,a,b,c,14,9,0xc33707d6);
		md5round_g(c,d,a,b,3,14,0xf4d50d87);
		md5round_g(b,c,d,a,8,20,0x455a14ed);
	md5round_g(a,b,c,d,13,5,0xa9e3e905);
		md5round_g(d,a,b,c,2,9,0xfcefa3f8);
		md5round_g(c,d,a,b,7,14,0x676f02d9);
		md5round_g(b,c,d,a,12,20,0x8d2a4c8a);

	md5round_h(a,b,c,d,5,4,0xfffa3942);
		md5round_h(d,a,b,c,8,11,0x8771f681);
		md5round_h(c,d,a,b,11,16,0x6d9d6122);
		md5round_h(b,c,d,a,14,23,0xfde5380c);
	md5round_h(a,b,c,d,1,4,0xa4beea44);
		md5round_h(d,a,b,c,4,11,0x4bdecfa9);
		md5round_h(c,d,a,b,7,16,0xf6bb4b60);
		md5round_h(b,c,d,a,10,23,0xbebfbc70);
	md5round_h(a,b,c,d,13,4,0x289b7ec6);
		md5round_h(d,a,b,c,0,11,0xeaa127fa);
		md5round_h(c,d,a,b,3,16,0xd4ef3085);
		md5round_h(b,c,d,a,6,23,0x04881d05);
	md5round_h(a,b,c,d,9,4,0xd9d4d039);
		md5round_h(d,a,b,c,12,11,0xe6db99e5);
		md5round_h(c,d,a,b,15,16,0x1fa27cf8);
		md5round_h(b,c,d,a,2,23,0xc4ac5665);

	md5round_i(a,b,c,d,0,6,0xf4292244);
		md5round_i(d,a,b,c,7,10,0x432aff97);
		md5round_i(c,d,a,b,14,15,0xab9423a7);
		md5round_i(b,c,d,a,5,21,0xfc93a039);
	md5round_i(a,b,c,d,12,6,0x655b59c3);
		md5round_i(d,a,b,c,3,10,0x8f0ccc92);
		md5round_i(c,d,a,b,10,15,0xffeff47d);
		md5round_i(b,c,d,a,1,21,0x85845dd1);
	md5round_i(a,b,c,d,8,6,0x6fa87e4f);
		md5round_i(d,a,b,c,15,10,0xfe2ce6e0);
		md5round_i(c,d,a,b,6,15,0xa3014314);
		md5round_i(b,c,d,a,13,21,0x4e0811a1);
	md5round_i(a,b,c,d,4,6,0xf7537e82);
		md5round_i(d,a,b,c,11,10,0xbd3af235);
		md5round_i(c,d,a,b,2,15,0x2ad7d2bb);
		md5round_i(b,c,d,a,9,21,0xeb86d391);
	
	//////////////////////////////////////////////////////////////////////////////
	//STEP 5: Output
	a += 0x67452301;
	b += 0xefcdab89;
	c += 0x98badcfe;
	d += 0x10325476;
	
	unsigned int xb[4] = {a, b, c, d};
	for(int i=0; i<4; i++)
	{
		int base=i*4;
		out[base] = xb[i] & 0xff; xb[i] >>= 8;
		out[base+1] = xb[i] & 0xff; xb[i] >>= 8;
		out[base+2] = xb[i] & 0xff; xb[i] >>= 8;
		out[base+3] = xb[i] & 0xff; xb[i] >>= 8;
	}
}

//Do an MD5 hash, with the following contraints:
//	pin is writable
//	pin != pout
//	pin, pout are never cast to char* (compiler bug)
void __device__ DoMD5HashFast(unsigned int* buf, unsigned int* pout, int length)
{
	//////////////////////////////////////////////////////////////////////////////
	//STEP 1: Copy buffer and append padding bits
	//Generate the string
	for(int j=ceil(length/4.0f); j<16; j++)				//Zero the end
		buf[j] = 0;
	unsigned char* p=(unsigned char*)buf;				//Byte ptr to buffer
	p[length]=0x80;										//Add padding
	for(int i=length+1; i%4 != 0; i++)
		p[i] = 0;
	
	//////////////////////////////////////////////////////////////////////////////
	//STEP 2: Append Length
	unsigned int xlen = length * 8;

	//////////////////////////////////////////////////////////////////////////////
	//STEP 3: Initialize Constants
	//Variable declarations
	unsigned int a,b,c,d;								//Temp variables for MD5 computation
	a = 0x67452301;
	b = 0xefcdab89;
	c = 0x98badcfe;
	d = 0x10325476;

	//////////////////////////////////////////////////////////////////////////////
	//STEP 4: MD5 Rounds
	fastmd5round_f(a,b,c,d,buf[0],7,0xd76aa478);
		fastmd5round_f(d,a,b,c,buf[1],12,0xe8c7b756);
		fastmd5round_f(c,d,a,b,buf[2],17,0x242070db);
		fastmd5round_f(b,c,d,a,buf[3],22,0xc1bdceee);
	fastmd5round_f(a,b,c,d,buf[4],7,0xf57c0faf);
		fastmd5round_f(d,a,b,c,buf[5],12,0x4787c62a);
		fastmd5round_f(c,d,a,b,buf[6],17,0xa8304613);
		fastmd5round_f(b,c,d,a,buf[7],22,0xfd469501);
	fastmd5round_f(a,b,c,d,buf[8],7,0x698098d8);
		fastmd5round_f(d,a,b,c,buf[9],12,0x8b44f7af);
		fastmd5round_f(c,d,a,b,buf[10],17,0xffff5bb1);
		fastmd5round_f(b,c,d,a,buf[11],22,0x895cd7be);
	fastmd5round_f(a,b,c,d,buf[12],7,0x6b901122);
		fastmd5round_f(d,a,b,c,buf[13],12,0xfd987193);
		fastmd5round_f(c,d,a,b,xlen,17,0xa679438e);
		fastmd5round_f(b,c,d,a,0,22,0x49b40821);

	fastmd5round_g(a,b,c,d,buf[1],5,0xf61e2562);
		fastmd5round_g(d,a,b,c,buf[6],9,0xc040b340);
		fastmd5round_g(c,d,a,b,buf[11],14,0x265e5a51);
		fastmd5round_g(b,c,d,a,buf[0],20,0xe9b6c7aa);
	fastmd5round_g(a,b,c,d,buf[5],5,0xd62f105d);
		fastmd5round_g(d,a,b,c,buf[10],9,0x02441453);
		fastmd5round_g(c,d,a,b,0,14,0xd8a1e681);
		fastmd5round_g(b,c,d,a,buf[4],20,0xe7d3fbc8);
	fastmd5round_g(a,b,c,d,buf[9],5,0x21e1cde6);
		fastmd5round_g(d,a,b,c,xlen,9,0xc33707d6);
		fastmd5round_g(c,d,a,b,buf[3],14,0xf4d50d87);
		fastmd5round_g(b,c,d,a,buf[8],20,0x455a14ed);
	fastmd5round_g(a,b,c,d,buf[13],5,0xa9e3e905);
		fastmd5round_g(d,a,b,c,buf[2],9,0xfcefa3f8);
		fastmd5round_g(c,d,a,b,buf[7],14,0x676f02d9);
		fastmd5round_g(b,c,d,a,buf[12],20,0x8d2a4c8a);

	fastmd5round_h(a,b,c,d,buf[5],4,0xfffa3942);
		fastmd5round_h(d,a,b,c,buf[8],11,0x8771f681);
		fastmd5round_h(c,d,a,b,buf[11],16,0x6d9d6122);
		fastmd5round_h(b,c,d,a,xlen,23,0xfde5380c);
	fastmd5round_h(a,b,c,d,buf[1],4,0xa4beea44);
		fastmd5round_h(d,a,b,c,buf[4],11,0x4bdecfa9);
		fastmd5round_h(c,d,a,b,buf[7],16,0xf6bb4b60);
		fastmd5round_h(b,c,d,a,buf[10],23,0xbebfbc70);
	fastmd5round_h(a,b,c,d,buf[13],4,0x289b7ec6);
		fastmd5round_h(d,a,b,c,buf[0],11,0xeaa127fa);
		fastmd5round_h(c,d,a,b,buf[3],16,0xd4ef3085);
		fastmd5round_h(b,c,d,a,buf[6],23,0x04881d05);
	fastmd5round_h(a,b,c,d,buf[9],4,0xd9d4d039);
		fastmd5round_h(d,a,b,c,buf[12],11,0xe6db99e5);
		fastmd5round_h(c,d,a,b,0,16,0x1fa27cf8);
		fastmd5round_h(b,c,d,a,buf[2],23,0xc4ac5665);

	fastmd5round_i(a,b,c,d,buf[0],6,0xf4292244);
		fastmd5round_i(d,a,b,c,buf[7],10,0x432aff97);
		fastmd5round_i(c,d,a,b,xlen,15,0xab9423a7);
		fastmd5round_i(b,c,d,a,buf[5],21,0xfc93a039);
	fastmd5round_i(a,b,c,d,buf[12],6,0x655b59c3);
		fastmd5round_i(d,a,b,c,buf[3],10,0x8f0ccc92);
		fastmd5round_i(c,d,a,b,buf[10],15,0xffeff47d);
		fastmd5round_i(b,c,d,a,buf[1],21,0x85845dd1);
	fastmd5round_i(a,b,c,d,buf[8],6,0x6fa87e4f);
		fastmd5round_i(d,a,b,c,0,10,0xfe2ce6e0);
		fastmd5round_i(c,d,a,b,buf[6],15,0xa3014314);
		fastmd5round_i(b,c,d,a,buf[13],21,0x4e0811a1);
	fastmd5round_i(a,b,c,d,buf[4],6,0xf7537e82);
		fastmd5round_i(d,a,b,c,buf[11],10,0xbd3af235);
		fastmd5round_i(c,d,a,b,buf[2],15,0x2ad7d2bb);
		fastmd5round_i(b,c,d,a,buf[9],21,0xeb86d391);
	
	//////////////////////////////////////////////////////////////////////////////
	//STEP 5: Output
	a += 0x67452301;
	b += 0xefcdab89;
	c += 0x98badcfe;
	d += 0x10325476;
	
	pout[0] = a;
	pout[1] = b;
	pout[2] = c;
	pout[3] = d;
}

int __device__ istrcat(unsigned int* dest, int destlen, unsigned int* src, int srclen)
{
	//Concatenate the strings (sliding window processing one source dword at a time)
	int base = floor(destlen/4.0f);
	int n = (destlen % 4) * 8;
	int fmn = 32 - n;
	unsigned int keepmask = 0xffffffff >> fmn;
	unsigned int rmask = 0xffffffff >> n;
	for(int i=0; i<=ceilf(srclen / 4.0f); i++)
	{
		int d = dest[base];
		int s = src[i];
		d &= keepmask;
		d |= ((s & rmask) << n);
		dest[base] = d;
		dest[base+1] = s >> fmn;
		base++;
	}
	
	//Return new length
	return destlen + srclen;
}

/*!
	@brief CUDA implementation of md5crypt	
	Thread-per-block requirement: minimum 64
	
	@param gtarget Target value (four ints, little endian)
	@param gstart Start index in charset (array of 32 ints, data is left aligned, unused values are at right)
	@param gsalt Salt
	@param status Set to true by a thread which succeeds in cracking the hash
	@param output Set to the collision by a thread which succeeds in cracking the hash
	@param base Length of the character set (passed in texCharset texture)
	@param len Length of valid data in gstart
	@param saltlen Length of salt
	@param hashcount Number of hashes being tested (not used)
 */
extern "C" __global__ void md5cryptKernel(int* gtarget, int* gstart, char* gsalt, char* status, char* output, int base, int len, int saltlen, int hashcount)
{
	//Get our position in the grid
	int index = (blockDim.x * blockIdx.x) + threadIdx.x;
	
	//Cache charset in shmem
	__shared__ char charset[256];
	if(threadIdx.x < ceil((float)base / 4))
	{
		int* ccs = (int*)&charset[0];
		ccs[threadIdx.x] = tex1Dfetch(texCharset, threadIdx.x);
	}
	
	//Cache start value
	__shared__ int start[32];
	if(threadIdx.x < len)
		start[threadIdx.x] = gstart[threadIdx.x];
		
	//Cache salt
	__shared__ unsigned int isalt[3];
	if(saltlen > 12)			//md5crypt is defined as having salt up to 64 bits max
		return;
	unsigned char* salt = reinterpret_cast<unsigned char*>(isalt);
	if(threadIdx.x < saltlen)
		salt[threadIdx.x] = gsalt[threadIdx.x];
	
	//Cache target value
	__shared__ int target[4];
	if(threadIdx.x < 4)
		target[threadIdx.x] = gtarget[threadIdx.x];
	
	//Wait for all cache filling to finish
	__syncthreads();
	
	//Calculate the value we're hashing
	int pos[20];
	int guessend = len-1;
	for(int i=0; i<guessend; i++)
		pos[i] = start[i];
	pos[guessend] = start[guessend] + index;
	for(int i=guessend; i>=0; i--)
	{
		//No carry? Quit
		if(pos[i] < base)
			break;
		
		//Split carry from digit
		int digit = pos[i] % base;
		int carry = pos[i] - digit;
		
		//Save digit, push carry over
		pos[i] = digit;
		if(i >= 1)
			pos[i-1] += carry/base;
	}
	unsigned int iin[8];
	unsigned char* in = reinterpret_cast<unsigned char*>(iin);
	{
		for(int i=0; i<len; i++)
			in[i] = charset[pos[i]];
	}
	
	//|| is defined as concatenation
	//foo := MD5(password || salt || password);
	unsigned int ifoo[16];
	unsigned char* foo = reinterpret_cast<unsigned char*>(ifoo);
	int foolen=0;
	{
		//Copy password		
		int base = floor(len/4.0f);
		for(int i=0; i<=base; i++)
			ifoo[i] = iin[i];
		foolen = len;
		
		//Append the salt and password
		foolen = istrcat(ifoo, foolen, isalt, saltlen);
		foolen = istrcat(ifoo, foolen, iin, len);
		
		//Do the hash
		DoMD5Hash(ifoo,ifoo,foolen);
	}
	
	// extend(str, len) is str repeated to fill len bytes.
	//  bar := password || "$1$" || salt || extend(foo, length(password));
	unsigned int ibar[16];
	unsigned char* bar = reinterpret_cast<unsigned char*>(ibar);
	int barlen = 0;
	for(int i=0; i<ceilf(len); i++)	//copy in dword blocks, extra (if any) will be overwritten by the $1$ text
		ibar[i] = iin[i];
	barlen += len;
	bar[barlen++]='$';
	bar[barlen++]='1';
	bar[barlen++]='$';
	barlen = istrcat(ibar, barlen, isalt, saltlen);
	for(int i=0; i<len; i++)		//extend(foo, inlen)
		bar[barlen+i] = foo[i%foolen];
	barlen += len;
	
	//Pad with alternating nulls and initial characters
	int i = len;	
	while(i > 0)
	{
		if(i % 2)
			bar[barlen++] = '\0';	//append a null, not an ascii '0' or nothing at all!
		else
			bar[barlen++] = in[0];
		i >>= 1;
	}
	
	//TODO: Cache some strings, but don't use more than 32 registers.
	//Since we can run up to 4x 128 threads on a computer 1.3 device with 16K of shmem, we cannot
	//use more than 4K of shmem per block.
		
	//baz := MD5(bar);
	unsigned int ibaz[16];
	DoMD5Hash(ibar,ibaz,barlen);
	
	//ifnz(i, s) is defined as (i==0) ? "" : s
	unsigned int itbaz[16];
	itbaz[0] = ibaz[0];
	itbaz[1] = ibaz[1];
	itbaz[2] = ibaz[2];
	itbaz[3] = ibaz[3];
	i=0;
	int lo4 = ceilf(len/4.0f);
	while(i<1000)
	{
		//baz := MD5(baz || ifnz(i mod 3, salt) || ifnz(i mod 7, password) || password);
		int tbazlen = 16;
		if(i%3 != 0)
		{
			//Append salt, knowing that itbaz is aligned nicely so no shifting is needed
			itbaz[4] = isalt[0];
			itbaz[5] = isalt[1];
			itbaz[6] = isalt[2];
			tbazlen = 16 + saltlen;
			
			//Append password
			if(i%7 != 0)
				tbazlen = istrcat(itbaz, tbazlen, iin, len);
		}
		else if(i%7 != 0)
		{
			//Append password, knowing that itbaz is aligned nicely so no shifting is needed
			//istrcat(itbaz, tbazlen, iin, len);
			itbaz[4] = iin[0];
			itbaz[5] = iin[1];
			itbaz[6] = iin[2];
			itbaz[7] = iin[3];
			itbaz[8] = iin[4];
			itbaz[9] = iin[5];
			itbaz[10] = iin[6];
			tbazlen = 16 + len;
		}
		tbazlen = istrcat(itbaz, tbazlen, iin, len);
		DoMD5HashFast(itbaz,ibaz,tbazlen);
		
		i++;
		
		//baz := MD5(password || ifnz(i mod 3, salt) || ifnz(i mod 7, password) || baz);
		for(int j=0; j<lo4; j++)
			itbaz[j] = iin[j];
		tbazlen = len;
		if(i%3 != 0)
			tbazlen = istrcat(itbaz, tbazlen, isalt, saltlen);
		if(i%7 != 0)
			tbazlen = istrcat(itbaz, tbazlen, iin, len);
		tbazlen = istrcat(itbaz, tbazlen, ibaz, 16);
		DoMD5HashFast(itbaz,itbaz,tbazlen);
		
		i++;
	}
	
	//Test result
	if(itbaz[0] != target[0])
		return;
	if(itbaz[1] != target[1])
		return;
	if(itbaz[2] != target[2])
		return;
	if(itbaz[3] != target[3])
		return;
	
	//Match if we got to here. Save result.
	*status = 1;
	for(int j=0; j<len; j++)
		output[j] = in[j];
}
