#include "hip/hip_runtime.h"
/******************************************************************************
*                                                                             *
* Distributed Hash Cracker v3.0                                               *
*                                                                             *
* Copyright (c) 2009 RPISEC.                                                  *
* All rights reserved.                                                        *
*                                                                             *
* Redistribution and use in source and binary forms, with or without modifi-  *
* cation, are permitted provided that the following conditions are met:       *
*                                                                             *
*    * Redistributions of source code must retain the above copyright notice  *
*      this list of conditions and the following disclaimer.                  *
*                                                                             *
*    * Redistributions in binary form must reproduce the above copyright      *
*      notice, this list of conditions and the following disclaimer in the    *
*      documentation and/or other materials provided with the distribution.   *
*                                                                             *
*    * Neither the name of RPISEC nor the names of its contributors may be    *
*      used to endorse or promote products derived from this software without *
*      specific prior written permission.                                     *
*                                                                             *
* THIS SOFTWARE IS PROVIDED BY RPISEC "AS IS" AND ANY EXPRESS OR IMPLIED      *
* WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF        *
* MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN     *
* NO EVENT SHALL RPISEC BE HELD LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,  *
* SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED    *
* TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR      *
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF      *
* LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING        *
* NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS          *
* SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.                *
*                                                                             *
******************************************************************************/

/*!
	@file md4_kernel.cu
	
	@brief CUDA implementation of MD4
 */

//Textures
texture<int, 1, hipReadModeElementType> texCharset;

//Fake-array macros
#define AddPadding(num) case num: \
		buf##num = (buf##num & ~paddmask) | padding; \
		break
#define InitGuess(num, a,b,c,d)				\
		{										\
			/* Calculate the four indices */	\
			LstartInit(lstart3, a);				\
			LstartInit(lstart2, b);				\
			LstartInit(lstart1, c);				\
			LstartInit(lstart0, d);				\
			/* Pack four elements into the int (if we exceed length, padding will overwrite the garbage) */	\
			buf##num = 							\
			(charset[lstart3] << 24) | 			\
			(charset[lstart2] << 16) | 			\
			(charset[lstart1] << 8) |  			\
			 charset[lstart0];					\
		}
		 
#define LstartInit(ls, num)								\
		{												\
			/* Get initial value and apply carry-in */	\
			ls = carry + start[num]; 					\
			/* Rightmost value? Bump by index */		\
			if(num == lm1) 								\
				ls += index;							\
			/* Carry out */								\
			if(ls >= base && num<len)					\
			{											\
				/* Calculate carry */					\
				carry = ls / base;						\
				/* Update this digit */					\
				ls %= base;								\
			}											\
			else										\
				carry = 0;								\
		}
		
#define SaveOutput(num) case num:					\
		po[num] = buf##num;

/*!
	@brief CUDA implementation of MD4
	
	Thread-per-block requirement: minimum 64
	
	@param gtarget Target value (four ints, little endian)
	@param gstart Start index in charset (array of 32 ints, data is left aligned, unused values are at right)
	@param gsalt Salt (not used)
	@param status Set to true by a thread which succeeds in cracking the hash
	@param output Set to the collision by a thread which succeeds in cracking the hash
	@param base Length of the character set (passed in texCharset texture)
	@param len Length of valid data in gstart
	@param saltlen Length of salt (not used)
	@param hashcount Number of hashes being tested (not used)
 */
extern "C" __global__ void md4Kernel(int* gtarget, int* gstart, char* gsalt, char* status, char* output, int base, int len, int saltlen, int hashcount)
{
	//Get our position in the grid
	int index = (blockDim.x * blockIdx.x) + threadIdx.x;
	
	//Cache charset in shmem
	__shared__ char charset[256];
	if(threadIdx.x < ceil((float)base / 4))
	{
		int* ccs = (int*)&charset[0];
		ccs[threadIdx.x] = tex1Dfetch(texCharset, threadIdx.x);
	}
	
	//Cache start value
	__shared__ int start[32];
	if(threadIdx.x < len)
		start[threadIdx.x] = gstart[threadIdx.x];
	
	//Cache target value
	__shared__ int target[4];
	if(threadIdx.x < 4)
		target[threadIdx.x] = gtarget[threadIdx.x];
	
	//Wait for all cache filling to finish
	__syncthreads();
	
	//Guess generation and round structure
	#include "md4ntlm_kernel_core.h"
	
	//Check results
	if(target[0] == a && target[1] == b && target[2] == c && target[3] == d)
	{
		*status = 1;
		
		int* po = (int*)output;
		switch(len / 4)
		{
			SaveOutput(7);
			SaveOutput(6);
			SaveOutput(5);
			SaveOutput(4);
			SaveOutput(3);
			SaveOutput(2);
			SaveOutput(1);
			SaveOutput(0);
		}
	}
}

/*!
	@brief CUDA implementation of MD4 with meet-in-the-middle attack
	
	Thread-per-block requirement: minimum 64
	
	@param gtarget Target value (four ints, little endian). The last seven sub-rounds have been reversed by this point.
	@param gstart Start index in charset (array of 32 ints, data is left aligned, unused values are at right)
	@param gsalt Salt (not used)
	@param status Set to true by a thread which succeeds in cracking the hash
	@param output Set to the collision by a thread which succeeds in cracking the hash
	@param base Length of the character set (passed in texCharset texture)
	@param len Length of valid data in gstart
	@param saltlen Length of salt (not used)
	@param hashcount Number of hashes being tested (not used)
 */
extern "C" __global__ void md4_fastKernel(int* gtarget, int* gstart, char* gsalt, char* status, char* output, int base, int len, int saltlen, int hashcount)
{
	//Get our position in the grid
	int index = (blockDim.x * blockIdx.x) + threadIdx.x;
	
	//Cache charset in shmem
	__shared__ char charset[256];
	if(threadIdx.x < ceil((float)base / 4))
	{
		int* ccs = (int*)&charset[0];
		ccs[threadIdx.x] = tex1Dfetch(texCharset, threadIdx.x);
	}
	
	//Cache start value
	__shared__ int start[32];
	if(threadIdx.x < len)
		start[threadIdx.x] = gstart[threadIdx.x];
	
	//Cache target value
	__shared__ int target[4];
	if(threadIdx.x < 4)
		target[threadIdx.x] = gtarget[threadIdx.x];
	
	//Wait for all cache filling to finish
	__syncthreads();
	
	//Guess generation and round structure
	#define MD4_MEET_IN_THE_MIDDLE
	#include "md4ntlm_kernel_core.h"
	
	//Check results
	if(target[0] == a && target[1] == b && target[2] == c && target[3] == d)
	{
		*status = 1;
		
		int* po = (int*)output;
		switch(len / 4)
		{
			SaveOutput(7);
			SaveOutput(6);
			SaveOutput(5);
			SaveOutput(4);
			SaveOutput(3);
			SaveOutput(2);
			SaveOutput(1);
			SaveOutput(0);
		}
	}
}

/*!
	@brief CUDA implementation of MD4 with batch cracking support
	
	Thread-per-block requirement: minimum 64
	
	@param gtarget Target value (four ints, little endian)
	@param gstart Start index in charset (array of 32 ints, data is left aligned, unused values are at right)
	@param gsalt Salt (not used)
	@param status Set to true by a thread which succeeds in cracking the hash
	@param output Set to the collision by a thread which succeeds in cracking the hash
	@param base Length of the character set (passed in texCharset texture)
	@param len Length of valid data in gstart
	@param saltlen Length of salt (not used)
	@param hashcount Number of hashes being tested
 */
extern "C" __global__ void md4BatchKernel(int* gtarget, int* gstart, char* gsalt, char* status, char* output, int base, int len, int saltlen, int hashcount)
{
	//Get our position in the grid
	int index = (blockDim.x * blockIdx.x) + threadIdx.x;
	
	//Cache charset in shmem
	__shared__ char charset[256];
	if(threadIdx.x < ceil((float)base / 4))
	{
		int* ccs = (int*)&charset[0];
		ccs[threadIdx.x] = tex1Dfetch(texCharset, threadIdx.x);
	}
	
	//Cache start value
	__shared__ int start[32];
	if(threadIdx.x < len)
		start[threadIdx.x] = gstart[threadIdx.x];
	
	//Cache target value
	__shared__ int target[4 * 128];
	if(threadIdx.x < 64)
	{
		int td = threadIdx.x;
		if(td < hashcount)
		{
			for(int i=0; i<4; i++)
				target[4*td + i] = gtarget[4*td + i];
		}
		if(td > 64)
		{
			td -= 64;
			if(td < hashcount)
			{
				for(int i=0; i<4; i++)
					target[4*td + i] = gtarget[4*td + i];
			}
		}
	}
	
	//Wait for all cache filling to finish
	__syncthreads();
	
	//Guess generation and round structure
	#include "md4ntlm_kernel_core.h"
	
	//Check results
	for(int i=0; i<hashcount; i++)
	{
		int* xtarget = target + (4*i);
		
		//Check results
		if(xtarget[0] == a && xtarget[1] == b && xtarget[2] == c && xtarget[3] == d)
		{
			status[i] = 1;
			
			int* po = (int*)output + (i*8);
			switch(len / 4)
			{
				SaveOutput(7);
				SaveOutput(6);
				SaveOutput(5);
				SaveOutput(4);
				SaveOutput(3);
				SaveOutput(2);
				SaveOutput(1);
				SaveOutput(0);
			}
		}
	}
}


/*!
	@brief CUDA implementation of MD4 with batch cracking support and cryptanalysis
	
	Thread-per-block requirement: minimum 64
	
	@param gtarget Target value (four ints, little endian)
	@param gstart Start index in charset (array of 32 ints, data is left aligned, unused values are at right)
	@param gsalt Salt (not used)
	@param status Set to true by a thread which succeeds in cracking the hash
	@param output Set to the collision by a thread which succeeds in cracking the hash
	@param base Length of the character set (passed in texCharset texture)
	@param len Length of valid data in gstart
	@param saltlen Length of salt (not used)
	@param hashcount Number of hashes being tested
 */
extern "C" __global__ void md4_fastBatchKernel(int* gtarget, int* gstart, char* gsalt, char* status, char* output, int base, int len, int saltlen, int hashcount)
{
	//Get our position in the grid
	int index = (blockDim.x * blockIdx.x) + threadIdx.x;
	
	//Cache charset in shmem
	__shared__ char charset[256];
	if(threadIdx.x < ceil((float)base / 4))
	{
		int* ccs = (int*)&charset[0];
		ccs[threadIdx.x] = tex1Dfetch(texCharset, threadIdx.x);
	}
	
	//Cache start value
	__shared__ int start[32];
	if(threadIdx.x < len)
		start[threadIdx.x] = gstart[threadIdx.x];
	
	//Cache target value
	__shared__ int target[4 * 128];
	if(threadIdx.x < 64)
	{
		int td = threadIdx.x;
		if(td < hashcount)
		{
			for(int i=0; i<4; i++)
				target[4*td + i] = gtarget[4*td + i];
		}
		if(td > 64)
		{
			td -= 64;
			if(td < hashcount)
			{
				for(int i=0; i<4; i++)
					target[4*td + i] = gtarget[4*td + i];
			}
		}
	}
	
	//Wait for all cache filling to finish
	__syncthreads();
	
	//Guess generation and round structure
	#define MD4_MEET_IN_THE_MIDDLE
	#include "md4ntlm_kernel_core.h"
	
	//Check results
	for(int i=0; i<hashcount; i++)
	{
		int* xtarget = target + (4*i);
		
		//Check results
		if(xtarget[0] == a && xtarget[1] == b && xtarget[2] == c && xtarget[3] == d)
		{
			status[i] = 1;
			
			int* po = (int*)output + (i*8);
			switch(len / 4)
			{
				SaveOutput(7);
				SaveOutput(6);
				SaveOutput(5);
				SaveOutput(4);
				SaveOutput(3);
				SaveOutput(2);
				SaveOutput(1);
				SaveOutput(0);
			}
		}
	}
}


