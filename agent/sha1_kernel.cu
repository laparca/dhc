#include "hip/hip_runtime.h"
/******************************************************************************
*                                                                             *
* Distributed Hash Cracker v3.0                                               *
*                                                                             *
* Copyright (c) 2009 RPISEC.                                                  *
* All rights reserved.                                                        *
*                                                                             *
* Redistribution and use in source and binary forms, with or without modifi-  *
* cation, are permitted provided that the following conditions are met:       *
*                                                                             *
*    * Redistributions of source code must retain the above copyright notice  *
*      this list of conditions and the following disclaimer.                  *
*                                                                             *
*    * Redistributions in binary form must reproduce the above copyright      *
*      notice, this list of conditions and the following disclaimer in the    *
*      documentation and/or other materials provided with the distribution.   *
*                                                                             *
*    * Neither the name of RPISEC nor the names of its contributors may be    *
*      used to endorse or promote products derived from this software without *
*      specific prior written permission.                                     *
*                                                                             *
* THIS SOFTWARE IS PROVIDED BY RPISEC "AS IS" AND ANY EXPRESS OR IMPLIED      *
* WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF        *
* MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN     *
* NO EVENT SHALL RPISEC BE HELD LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,  *
* SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED    *
* TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR      *
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF      *
* LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING        *
* NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS          *
* SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.                *
*                                                                             *
******************************************************************************/

/*!
	@file sha1_kernel.cu
	
	@brief CUDA implementation of SHA1
 */

//Textures
texture<int, 1, hipReadModeElementType> texCharset;

#define sha1_f1(b,c,d) ( (b & c) | (~b & d) )
#define sha1_f2(b,c,d) (b ^ c ^ d)
#define sha1_f3(b,c,d) ( (b & c) | (b & d) | (c & d) )
#define sha1_f4(b,c,d) (b ^ c ^ d)

#define ROTL(a,shamt) (((a) << shamt) | ((a) >> (32-shamt)))

#define bswap(x) ( (x & 0xFF)<<24 | (x&0xFF00) << 8 | (x&0xFF0000) >> 8 | (x&0xFF000000) >> 24  )

//Fake-array macros
#define AddPadding(num) case num: \
		w##num = (w##num & ~paddmask) | padding; \
		break
#define InitGuess(num, a,b,c,d)				\
		{										\
			/* Calculate the four indices */	\
			LstartInit(lstart3, a);				\
			LstartInit(lstart2, b);				\
			LstartInit(lstart1, c);				\
			LstartInit(lstart0, d);				\
			/* Pack four elements into the int (if we exceed length, padding will overwrite the garbage) */	\
			w##num = 							\
			(charset[lstart3] << 24) | 			\
			(charset[lstart2] << 16) | 			\
			(charset[lstart1] << 8) |  			\
			 charset[lstart0];					\
		}
		 
#define LstartInit(ls, num)								\
		{												\
			/* Get initial value and apply carry-in */	\
			ls = carry + start[num]; 					\
			/* Rightmost value? Bump by index */		\
			if(num == lm1) 								\
				ls += index;							\
			/* Carry out */								\
			if(ls >= base && num<len)					\
			{											\
				/* Calculate carry */					\
				carry = ls / base;						\
				/* Update this digit */					\
				ls %= base;								\
			}											\
			else										\
				carry = 0;								\
		}
		
#define PostShift() \
		e=d;				\
		d=c;				\
		c=ROTL(b,30);		\
		b=a;				\
		a=temp;				\
		
#define PreShift()	\
	{	\
	/* Set w[round] */ \
	w16 = ROTL(w13 ^ w8 ^ w2 ^ w0,1); \
	/* Shift Ws */\
	w0 = w1; \
	w1 = w2; \
	w2 = w3; \
	w3 = w4; \
	w4 = w5; \
	w5 = w6; \
	w6 = w7; \
	w7 = w8; \
	w8 = w9; \
	w9 = w10; \
	w10 = w11; \
	w11 = w12; \
	w12 = w13; \
	w13 = w14; \
	w14 = w15; \
	w15 = w16; \
	}
	
#define RoundFromBlock1() \
	PreShift(); \
	temp=ROTL(a,5) + sha1_f1(b,c,d) + e + w16 + 0x5A827999;	\
	PostShift();

#define RoundFromBlock2() \
	PreShift(); \
	temp=ROTL(a,5) + sha1_f2(b,c,d) + e + w16 + 0x6ED9EBA1; \
	PostShift();
	
#define RoundFromBlock3() \
	PreShift(); \
	temp=ROTL(a,5) + e + w16 + 0x8F1BBCDC; \
	temp += sha1_f3(b,c,d); \
	PostShift();
	
#define RoundFromBlock4() \
	PreShift(); \
	temp=ROTL(a,5) + sha1_f4(b,c,d) + e + w16 + 0xCA62C1D6; \
	PostShift();
	
#define SaveOutput(num) \
	case num:	\
		reinterpret_cast<int*>(output)[num] = w##num;
		
#define SaveOutputBatch(num) \
case num:	\
	po[num] = w##num;

/*!
	@brief CUDA implementation of SHA1	
	Thread-per-block requirement: minimum 64
	
	@param gtarget Target value (five ints, little endian)
	@param gstart Start index in charset (array of 32 ints, data is left aligned, unused values are at right)
	@param gsalt Salt (not used)
	@param status Set to true by a thread which succeeds in cracking the hash
	@param output Set to the collision by a thread which succeeds in cracking the hash
	@param base Length of the character set (passed in texCharset texture)
	@param len Length of valid data in gstart
	@param saltlen Length of salt (not used)
	@param hashcount Number of hashes being tested (not used)
 */
extern "C" __global__ void sha1Kernel(int* gtarget, int* gstart, char* gsalt, char* status, char* output, int base, int len, int saltlen, int hashcount)
{
	//Get our position in the grid
	int index = (blockDim.x * blockIdx.x) + threadIdx.x;
	
	//Cache charset in shmem
	__shared__ char charset[256];
	if(threadIdx.x < ceil((float)base / 4))
	{
		int* ccs = (int*)&charset[0];
		ccs[threadIdx.x] = tex1Dfetch(texCharset, threadIdx.x);
	}
	
	//Cache start value
	__shared__ int start[32];
	if(threadIdx.x < len)
		start[threadIdx.x] = gstart[threadIdx.x];
	
	//Cache target value (do byte-swap here)
	__shared__ int target[5];
	if(threadIdx.x < 5)
		target[threadIdx.x] = bswap(gtarget[threadIdx.x]);
	
	//Wait for all cache filling to finish
	__syncthreads();
	
	//Core round functions
	#include "sha1_kernel_core.h"
		
	//Test the output
	int* pt = (int*)target;
	if( (pt[0] == a) && (pt[1] == b) && (pt[2] == c) && (pt[3] == d) && (pt[4] == e))
	{
		//If we get here, we must be a match! Save the result and quit
		*status = 1;
		unsigned int lo4 = len/4;
		
		unsigned int lstart0=0, lstart1=0, lstart2=0, lstart3=0;
		unsigned int lm1 = len-1;
		unsigned int carry = 0;		//no initial carry-in
		switch(lo4)
		{
		case 7:
			InitGuess(7, 31,30,29,28);
		case 6:
			InitGuess(6, 27,26,25,24);
		case 5:
			InitGuess(5, 23,22,21,20);
		case 4:
			InitGuess(4, 19,18,17,16);
		case 3:
			InitGuess(3, 15,14,13,12);
		case 2:
			InitGuess(2, 11,10,9,8);
		case 1:
			InitGuess(1, 7,6,5,4);
		case 0:
		default:
			InitGuess(0, 3,2,1,0);
		}
		
		switch(lo4)
		{
		SaveOutput(7);
		SaveOutput(6);
		SaveOutput(5);
		SaveOutput(4);
		SaveOutput(3);	
		SaveOutput(2);
		SaveOutput(1);
		SaveOutput(0);
		}
	}
}

/*!
	@brief CUDA implementation of SHA1 with batch processing support
	Thread-per-block requirement: minimum 64
	
	@param gtarget Target value (five ints, little endian)
	@param gstart Start index in charset (array of 32 ints, data is left aligned, unused values are at right)
	@param gsalt Salt (not used)
	@param status Set to true by a thread which succeeds in cracking the hash
	@param output Set to the collision by a thread which succeeds in cracking the hash
	@param base Length of the character set (passed in texCharset texture)
	@param len Length of valid data in gstart
	@param saltlen Length of salt (not used)
	@param hashcount Number of hashes being tested (not used)
 */
extern "C" __global__ void sha1BatchKernel(int* gtarget, int* gstart, char* gsalt, char* status, char* output, int base, int len, int saltlen, int hashcount)
{
	//Get our position in the grid
	int index = (blockDim.x * blockIdx.x) + threadIdx.x;
	
	//Cache charset in shmem
	__shared__ char charset[256];
	if(threadIdx.x < ceil((float)base / 4))
	{
		int* ccs = (int*)&charset[0];
		ccs[threadIdx.x] = tex1Dfetch(texCharset, threadIdx.x);
	}
	
	//Cache start value
	__shared__ int start[32];
	if(threadIdx.x < len)
		start[threadIdx.x] = gstart[threadIdx.x];
	
	//Cache target value (do byte-swap here)
	
	__shared__ int target[5 * 128];
	if(threadIdx.x < 64)
	{
		int td = threadIdx.x;
		if(td < hashcount)
		{
			for(int i=0; i<5; i++)
				target[5*td + i] = bswap(gtarget[5*td + i]);
		}
		if(td > 64)
		{
			td -= 64;
			if(td < hashcount)
			{
				for(int i=0; i<5; i++)
					target[5*td + i] = bswap(gtarget[5*td + i]);
			}
		}
	}
	
	//Wait for all cache filling to finish
	__syncthreads();
	
	//Core round functions
	#include "sha1_kernel_core.h"
	
	//Test the output
	for(int i=0; i<hashcount; i++)
	{
		int* xtarget = target + (5*i);
		
		//Check results
		if(xtarget[0] == a && xtarget[1] == b && xtarget[2] == c && xtarget[3] == d && xtarget[4] == e)
		{
			status[i] = 1;
			
			unsigned int lo4 = len/4;
			
			unsigned int lstart0=0, lstart1=0, lstart2=0, lstart3=0;
			unsigned int lm1 = len-1;
			unsigned int carry = 0;		//no initial carry-in
			switch(lo4)
			{
			case 7:
				InitGuess(7, 31,30,29,28);
			case 6:
				InitGuess(6, 27,26,25,24);
			case 5:
				InitGuess(5, 23,22,21,20);
			case 4:
				InitGuess(4, 19,18,17,16);
			case 3:
				InitGuess(3, 15,14,13,12);
			case 2:
				InitGuess(2, 11,10,9,8);
			case 1:
				InitGuess(1, 7,6,5,4);
			case 0:
			default:
				InitGuess(0, 3,2,1,0);
			}
			
			int* po = (int*)output + (i*8);
			switch(lo4)
			{
			SaveOutputBatch(7);
			SaveOutputBatch(6);
			SaveOutputBatch(5);
			SaveOutputBatch(4);
			SaveOutputBatch(3);	
			SaveOutputBatch(2);
			SaveOutputBatch(1);
			SaveOutputBatch(0);
			}
		}
	}
}
