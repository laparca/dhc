#include "hip/hip_runtime.h"
/******************************************************************************
*                                                                             *
* Distributed Hash Cracker v3.0                                               *
*                                                                             *
* Copyright (c) 2009 RPISEC.                                                  *
* All rights reserved.                                                        *
*                                                                             *
* Redistribution and use in source and binary forms, with or without modifi-  *
* cation, are permitted provided that the following conditions are met:       *
*                                                                             *
*    * Redistributions of source code must retain the above copyright notice  *
*      this list of conditions and the following disclaimer.                  *
*                                                                             *
*    * Redistributions in binary form must reproduce the above copyright      *
*      notice, this list of conditions and the following disclaimer in the    *
*      documentation and/or other materials provided with the distribution.   *
*                                                                             *
*    * Neither the name of RPISEC nor the names of its contributors may be    *
*      used to endorse or promote products derived from this software without *
*      specific prior written permission.                                     *
*                                                                             *
* THIS SOFTWARE IS PROVIDED BY RPISEC "AS IS" AND ANY EXPRESS OR IMPLIED      *
* WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF        *
* MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN     *
* NO EVENT SHALL RPISEC BE HELD LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,  *
* SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED    *
* TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR      *
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF      *
* LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING        *
* NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS          *
* SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.                *
*                                                                             *
******************************************************************************/

/*!
	@file ntlm_kernel.cu
	
	@brief CUDA implementation of NTLM
 */

//Textures
texture<int, 1, hipReadModeElementType> texCharset;

//Fake-array macros
#define AddPadding(num) case num: \
		buf##num = (buf##num & ~paddmask) | padding; \
		break
#define InitGuess(num, a,b)				\
		{										\
			/* Calculate the four indices */	\
			LstartInit(lstart1, a);				\
			LstartInit(lstart0, b);				\
			/* Pack two elements into the int with null bytes in between (if we exceed length, padding will overwrite the garbage) */	\
			buf##num = 							\
			(charset[lstart1] << 16) | 			\
			 charset[lstart0];					\
		}
		 
#define LstartInit(ls, num)								\
		{												\
			/* Get initial value and apply carry-in */	\
			ls = carry + start[num]; 					\
			/* Rightmost value? Bump by index */		\
			if(num == lm1) 								\
				ls += index;							\
			/* Carry out */								\
			if(ls >= base && num<len)					\
			{											\
				/* Calculate carry */					\
				carry = ls / base;						\
				/* Update this digit */					\
				ls %= base;								\
			}											\
			else										\
				carry = 0;								\
		}
		
#define SaveOutput(num) case num:					\
		po[num] = buf##num;

/*!
	@brief CUDA implementation of NTLM
	
	Thread-per-block requirement: minimum 64
	
	@param gtarget Target value (four ints, little endian)
	@param gstart Start index in charset (array of 32 ints, data is left aligned, unused values are at right)
	@param gsalt Salt (not used)
	@param status Set to true by a thread which succeeds in cracking the hash
	@param output Set to the collision by a thread which succeeds in cracking the hash
	@param base Length of the character set (passed in texCharset texture)
	@param len Length of valid data in gstart
	@param saltlen Length of salt (not used)
	@param hashcount Number of hashes being tested (not used)
 */
extern "C" __global__ void ntlmKernel(int* gtarget, int* gstart, char* gsalt, char* status, char* output, int base, int len, int saltlen, int hashcount)
{
	//Get our position in the grid
	int index = (blockDim.x * blockIdx.x) + threadIdx.x;
	
	//Cache charset in shmem
	__shared__ char charset[256];
	if(threadIdx.x < ceil((float)base / 4))
	{
		int* ccs = (int*)&charset[0];
		ccs[threadIdx.x] = tex1Dfetch(texCharset, threadIdx.x);
	}
	
	//Cache start value
	__shared__ int start[32];
	if(threadIdx.x < len)
		start[threadIdx.x] = gstart[threadIdx.x];
	
	//Cache target value
	__shared__ int target[4];
	if(threadIdx.x < 4)
		target[threadIdx.x] = gtarget[threadIdx.x];
	
	//Wait for all cache filling to finish
	__syncthreads();
	
	//Guess generation and round structure
	#define NTLM
	#include "md4ntlm_kernel_core.h"
		
	//Check results
	if(target[0] == a && target[1] == b && target[2] == c && target[3] == d)
	{
		*status = 1;
		
		int pos[20];
		int guessend = len-1;
		for(int i=0; i<guessend; i++)
			pos[i] = start[i];
		pos[guessend] = start[guessend] + index;
		for(int i=guessend; i>=0; i--)
		{
			//No carry? Quit
			if(pos[i] < base)
				break;
			
			//Split carry from digit
			int digit = pos[i] % base;
			int carry = pos[i] - digit;
			
			//Save digit, push carry over
			pos[i] = digit;
			if(i >= 1)
				pos[i-1] += carry/base;
		}
		
		//Convert to ASCII and save
		for(int i=0; i<len; i++)
			output[i] = charset[pos[i]];
	}
}

/*!
	@brief CUDA implementation of NTLM with batch processing support
	
	Thread-per-block requirement: minimum 64
	
	@param gtarget Target value (four ints, little endian)
	@param gstart Start index in charset (array of 32 ints, data is left aligned, unused values are at right)
	@param gsalt Salt (not used)
	@param status Set to true by a thread which succeeds in cracking the hash
	@param output Set to the collision by a thread which succeeds in cracking the hash
	@param base Length of the character set (passed in texCharset texture)
	@param len Length of valid data in gstart
	@param saltlen Length of salt (not used)
	@param hashcount Number of hashes being tested
 */
extern "C" __global__ void ntlmBatchKernel(int* gtarget, int* gstart, char* gsalt, char* status, char* output, int base, int len, int saltlen, int hashcount)
{
	//Get our position in the grid
	int index = (blockDim.x * blockIdx.x) + threadIdx.x;
	
	//Cache charset in shmem
	__shared__ char charset[256];
	if(threadIdx.x < ceil((float)base / 4))
	{
		int* ccs = (int*)&charset[0];
		ccs[threadIdx.x] = tex1Dfetch(texCharset, threadIdx.x);
	}
	
	//Cache start value
	__shared__ int start[32];
	if(threadIdx.x < len)
		start[threadIdx.x] = gstart[threadIdx.x];
	
	//Cache target value
	__shared__ int target[4 * 128];
	if(threadIdx.x < 64)
	{
		int td = threadIdx.x;
		if(td < hashcount)
		{
			for(int i=0; i<4; i++)
				target[4*td + i] = gtarget[4*td + i];
		}
		if(td > 64)
		{
			td -= 64;
			if(td < hashcount)
			{
				for(int i=0; i<4; i++)
					target[4*td + i] = gtarget[4*td + i];
			}
		}
	}
	
	//Wait for all cache filling to finish
	__syncthreads();
	
	//Guess generation and round structure
	#define NTLM
	#include "md4ntlm_kernel_core.h"
		
	//Check results
	for(int h=0; h<hashcount; h++)
	{
		int* xtarget = target + (4*h);
		
		//Check results
		if(xtarget[0] == a && xtarget[1] == b && xtarget[2] == c && xtarget[3] == d)
		{
			status[h] = 1;
			int spos = h*32;
		
			int pos[20];
			int guessend = len-1;
			for(int i=0; i<guessend; i++)
				pos[i] = start[i];
			pos[guessend] = start[guessend] + index;
			for(int i=guessend; i>=0; i--)
			{
				//No carry? Quit
				if(pos[i] < base)
					break;
				
				//Split carry from digit
				int digit = pos[i] % base;
				int carry = pos[i] - digit;
				
				//Save digit, push carry over
				pos[i] = digit;
				if(i >= 1)
					pos[i-1] += carry/base;
			}
			
			//Convert to ASCII and save
			for(int i=0; i<len; i++)
				output[i+spos] = charset[pos[i]];
		}
	}
}
